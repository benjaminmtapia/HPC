
#include <hip/hip_runtime.h>
# include <stdio.h>
# include <stdlib.h>
# include <math.h>
# include <ctype.h>
# include <unistd.h>
# include <string.h>
# include <sys/time.h>

// Device code
__global__ void VecAdd(float* A, float* B, float* C, int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N)
        C[i] = A[i] + B[i];
}
            
// Host code
int main()
{
    int N = 512;
    size_t size = N * sizeof(float);

    // Allocate input vectors h_A and h_B in host memory
    float* h_A = (float*)malloc(sizeof(float)*N);
    float* h_B = (float*)malloc(sizeof(float)*N);
    float* h_C = (float*)malloc(sizeof(float)*N);

    // Initialize input vectors
    for(int i = 0; i < N; i++){
        h_A[i]=cos(i);
        h_B[i]=sin(i);
        h_C[i]=0;
    }

    // Allocate vectors in device memory
    float* d_A;
    hipMalloc(&d_A, size);
    float* d_B;
    hipMalloc(&d_B, size);
    float* d_C;
    hipMalloc(&d_C, size);

    // Copy vectors from host memory to device memory
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Invoke kernel
    int threadsPerBlock = 256;
    int blocksPerGrid =
            (N + threadsPerBlock - 1) / threadsPerBlock;
    VecAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

    // Copy result from device memory to host memory
    // h_C contains the result in host memory
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    for(int i = 0; i < N; i++){
        printf("%f ",h_C[i]);
    }
    printf("\n");
            
    // Free host memory
return 0;
}