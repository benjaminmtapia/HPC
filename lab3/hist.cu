#include "hip/hip_runtime.h"
#include "functions.h"
int D =0;

__global__ void histgmem(unsigned short int* I, int M, int N, int* H){
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    
    
    int npixels = M*N;
    int nthreads = gridDim.x * blockDim.x;
    if(nthreads<npixels){
        //some threads will add twice to the global histogram
        atomicAdd(&(H[I[i]]),1);
        //we calculate the left threads
        int left = npixels - nthreads;
        int newIndex = left - i;
        if(newIndex>=npixels){
            atomicAdd(&(H[I[i]]),1);
        }
    }
    else if(nthreads>npixels){
        if(i<npixels){
            atomicAdd(&(H[I[i]]),1);
        }
    }
    else{
        atomicAdd(&(H[I[i]]),1);
    }
    __syncthreads();


}
__global__ void histogram(unsigned short int* I, int M, int N, int* H){
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    atomicAdd(&(H[I[i]]),1);
    __syncthreads();


}
__global__ void histsmem(unsigned short int* I, int M, int N, int* H){
    int T = gridDim.x * blockDim.x; //blocksize
    int i = threadIdx.x; //local id
    int j = blockIdx.x*T + i; //global id
    __shared__ int hist[256];

    __syncthreads();
    if(j<M*N)
    atomicAdd(&(hist[I[j]]),1);
    __syncthreads();
    //add each local hist to global histogram, but only thread 0 will do this per block
    if(threadIdx.x == 0){
        for(int i = 0; i <=255;i++){
            atomicAdd(&(H[i]),hist[i]);
        }
    }
    

}

__host__ int main(int argc, char** argv){
    char* fileIn;
    int M = 0,N,blockSize;
    char* fileOut;
    int c;
    if(argc<10){
        printf("faltan parámetros\n");
        exit(1);
    }

    while (( (c = getopt(argc, argv, "i:m:n:o:t:d:")) != -1)){
        switch (c)
        {
        case 'i':
            fileIn = optarg;
            break;

        case 'm':
            sscanf(optarg,"%d",&M);
            break;

        case 'n':
            sscanf(optarg,"%d",&N);
            break;

        case 'o':
            fileOut = optarg;
            break;
        
        case 't':
            sscanf(optarg,"%d",&blockSize);
            break;

        case 'd':
            sscanf(optarg,"%d",&D);
            if(D!=0 && D!=1){
                printf("parametros incorrectos para D, solo puede ser 0 o 1\n");
                exit(1);
            }
            break;
        }
    }
   
    unsigned short int *image;
    int imageSize = M*N;
    image = readFile(fileIn,M,N);
    int *H = (int*) malloc(sizeof(int)*256);
    
    for(int i = 0; i<=255;i++){
        H[i]=0;
    }
    //device variables
    unsigned short int* image_d;
    int *hist_d;
    //memory allocation and copy into device memory
    hipMalloc(&image_d, imageSize*sizeof(unsigned short int));
    hipMalloc(&hist_d, 256*sizeof(int));
    hipMemcpy(image_d,image,imageSize*sizeof(unsigned short int), hipMemcpyHostToDevice);
    hipMemcpy(hist_d,H,256*sizeof(int), hipMemcpyHostToDevice);
    int gridSize = (int)ceil(imageSize/blockSize);
    histogram<<<gridSize, blockSize>>>(image_d, M, N, hist_d);
    hipDeviceSynchronize();
    int * histo = (int*)malloc(sizeof(int)*256);
    hipMemcpy(H,hist_d,256*sizeof(int),hipMemcpyDeviceToHost);

    //histogramming(image,M,N,H);

   writeFile(fileOut,H,256);
}