#include "hip/hip_runtime.h"
#include "functions.h"
# include <stdio.h>
# include <stdlib.h>
# include <math.h>
# include <ctype.h>
# include <unistd.h>
# include <string.h>
# include <sys/time.h>

int D =0;

__global__ void histgmem(unsigned short int* I, int M, int N, int* H){
    int i = blockDim.x * blockIdx.x + threadIdx.x;    
    int npixels = M*N;
    int nthreads = gridDim.x * blockDim.x;

    
    if(nthreads<npixels){
        //some threads will add twice to the global histogram
        atomicAdd(&(H[I[i]]),1);
        //we calculate the left threads
        int newIndex = nthreads+i+1;
        if(newIndex<npixels){
            atomicAdd(&(H[I[newIndex]]),1);
        }
    }
    else if(nthreads>=npixels){
        if(i<npixels){
            atomicAdd(&(H[I[i]]),1);
        }
    }
    __syncthreads();


}

__global__ void histsmem(unsigned short int* I, int M, int N, int* H, int T){
    int i = threadIdx.x; //local id
    int j = blockDim.x * blockIdx.x + threadIdx.x; //global id
    int npixels = M*N;
    int nthreads = gridDim.x * blockDim.x;
    __shared__ int hist[256];
    for(int i = 0; i<256;i++){
        hist[i]=0;
    }

    __syncthreads();

    if(nthreads>=npixels)
    atomicAdd(&(hist[I[j]]),1);

    else if(nthreads<npixels){
        atomicAdd(&(hist[I[j]]),1);
        int newIndex = nthreads+1+j;
        if(newIndex<npixels){
            atomicAdd(&(hist[I[newIndex]]),1);
        }
    }
    __syncthreads();
    //add each local hist to global histogram, but only thread 0 will do this per block
    if(i==0){
        for(int i = 0;i<256;i++)
        atomicAdd(&(H[i]),hist[i]);
    }
    __syncthreads();

}

__host__ int main(int argc, char** argv){
    char* fileIn;
    int M = 0,N,blockSize;
    char* fileOut;
    int c;
    if(argc<10){
        printf("faltan parámetros\n");
        exit(1);
    }

    while (( (c = getopt(argc, argv, "i:m:n:o:t:d:")) != -1)){
        switch (c)
        {
        case 'i':
            fileIn = optarg;
            break;

        case 'm':
            sscanf(optarg,"%d",&M);
            break;

        case 'n':
            sscanf(optarg,"%d",&N);
            break;

        case 'o':
            fileOut = optarg;
            break;
        
        case 't':
            sscanf(optarg,"%d",&blockSize);
            break;

        case 'd':
            sscanf(optarg,"%d",&D);
            if(D!=0 && D!=1){
                printf("parametros incorrectos para D, solo puede ser 0 o 1\n");
                exit(1);
            }
            break;
        }
    }
   
    unsigned short int *image;
    int imageSize = M*N;
    image = readFile(fileIn,M,N);
    int *H = (int*) malloc(sizeof(int)*256);
    int *H2 = (int*) malloc(sizeof(int)*256);
    
    for(int i = 0; i<=255;i++){
        H[i]=0;
        H2[i]=0;
    }

    //device variables
    unsigned short int* image_d;
    int *hist_d;
    int *hist_d2;

    //time variables
    hipEvent_t startGmem, startSmem, stopGmem, stopSmem;
    hipEventCreate(&startGmem);
    hipEventCreate(&stopGmem);
    hipEventCreate(&startSmem);
    hipEventCreate(&stopSmem);
    

    //memory allocation and copy into device memory
    hipMalloc(&image_d, imageSize*sizeof(unsigned short int));
    hipMalloc(&hist_d, 256*sizeof(int));
    hipMalloc(&hist_d2, 256*sizeof(int));

    hipMemcpy(image_d,image,imageSize*sizeof(unsigned short int), hipMemcpyHostToDevice);
    hipMemcpy(hist_d,H,256*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(hist_d2,H2,256*sizeof(int), hipMemcpyHostToDevice);

    int gridSize = (int)ceil(imageSize/blockSize);

    
    hipEventRecord(startGmem);
    histgmem<<<gridSize, blockSize>>>(image_d, M, N, hist_d);
    hipEventRecord(stopGmem);
    hipEventSynchronize(stopGmem);


    float timeinmillisecondsGlobal = 0;
    hipEventElapsedTime(&timeinmillisecondsGlobal, startGmem, stopGmem);


    hipEventRecord(startSmem);
    histsmem<<<gridSize, blockSize>>>(image_d, M, N, hist_d2,blockSize);
    hipEventRecord(stopSmem);
    hipEventSynchronize(stopSmem);
    hipDeviceSynchronize();

    float timeinmillisecondsShared = 0;
    hipEventElapsedTime(&timeinmillisecondsShared, startSmem, stopSmem);

    

    hipMemcpy(H,hist_d,256*sizeof(int),hipMemcpyDeviceToHost);
    hipMemcpy(H2,hist_d2,256*sizeof(int),hipMemcpyDeviceToHost);


    if(D==1)
    printHist(H,H2);

    printf("Global memory time: %f ms\n",timeinmillisecondsGlobal);
    printf("Shared memory time: %f ms\n",timeinmillisecondsShared);

    writeFile(fileOut,H,H2);
}